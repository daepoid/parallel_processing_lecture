#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
// 32 64 128
#define MAX 16 // matrix length
#define BLOCKS 8
#define EPOCH 100
#pragma warning(disable:4996)

__global__ void matrix_multiplication(double* C, double* A, double* B) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  double sum = 0.0;
  if (row < MAX && col < MAX) {
    for (int i = 0; i < MAX; i++) {
      sum += A[MAX * row + i] * B[MAX * i + col];
    }
  }
  C[MAX * row + col] = sum;
}

int main() {
  double A[MAX * MAX], B[MAX * MAX], C[MAX * MAX];
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start); // create start event
  hipEventCreate(&stop); // create stop event
  hipEventRecord(start, 0); // record start event

  for (int i = 0; i < MAX * MAX; i++) { // create A, B data
    A[i] = 1;
    B[i] = 1;
    C[i] = 0;
  }

  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
    double* cuda_A = 0;
    double* cuda_B = 0;
    double* cuda_C = 0;
    hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(double));
    hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(double));
    hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(double));

    hipMemcpy(cuda_A, A, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCKS, BLOCKS);
    dim3 dimGrid(MAX / BLOCKS, MAX / BLOCKS);

    hipEventCreate(&start); // �ð� ���
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrix_multiplication << < dimGrid, dimBlock >> > (cuda_C, cuda_A, cuda_B); // matrix multiplication
    //hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, cuda_C, MAX * MAX * sizeof(double), hipMemcpyDeviceToHost);

    if (t == 0) {
      for (int i = 0; i < MAX * MAX; i++) {
        if ((i + 1) % MAX == 0) {
          printf("%.1lf\n", C[i]);
        }
        else {
          printf("%.1lf ", C[i]);
        }
      }
    }

    time_avg += time;

    // �޸� ����
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
  }

  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
  return 0;
}
