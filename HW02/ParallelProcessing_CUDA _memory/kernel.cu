#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <hipblas.h>
#include <hipblas.h>
// 32 64 128
#define MAX 16 // matrix length
#define BLOCKS 16
#define EPOCH 100
#pragma warning(disable:4996)

#define BLOCK_SIZE 16

__global__ void MatMulKernel(double* A, double* B, double* C);

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}
// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
  Matrix Asub;
  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.stride = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(double* A, double* B, double* C) {
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0.0;
  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;
  // Loop over all the sub-matrices of A and B that are
  // required to compute Csub
  // Multiply each pair of sub-matrices together
  // and accumulate the results
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
    // Get sub-matrix Asub of A
    Matrix Asub = GetSubMatrix(A, blockRow, m);
    Matrix Bsub = GetSubMatrix(B, m, blockCol);
    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);
    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();
    // Multiply Asub and Bsub together
    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];
    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }
  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, Cvalue);
}

void MatMul(double* A, double* B, double* C) {
  // Load A and B to device memory
  double* cuda_A = 0;
  double* cuda_B = 0;
  double* cuda_C = 0;
  hipError_t err;

  hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(double));
  hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(double));
  hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(double));

  hipMemcpy(cuda_A, A, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(cuda_B, B, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(cuda_C, C, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(MAX / dimBlock.x, MAX / dimBlock.y);
  MatMulKernel << <dimGrid, dimBlock >> > (cuda_A, cuda_B, cuda_C);

  err = hipDeviceSynchronize();
  printf("Run kernel: %s\n", hipGetErrorString(err));
  // Read C from device memory
  err = hipMemcpy(C, cuda_C, MAX * MAX * sizeof(double), hipMemcpyDeviceToHost);
  printf("Copy C off of device: %s\n", hipGetErrorString(err));
  // Free device memory
  hipFree(cuda_A);
  hipFree(cuda_B);
  hipFree(cuda_C);
}

int main() {
  double A[MAX * MAX], B[MAX * MAX], C[MAX * MAX];

  for (int i = 0; i < MAX * MAX; i++) {
    A[i] = 1.0;
    B[i] = 1.0;
    C[i] = 0.0;
  }

  /*
  A.elements = (float*)malloc(MAX * MAX * sizeof(float));
  B.elements = (float*)malloc(MAX * MAX * sizeof(float));
  C.elements = (float*)malloc(MAX * MAX * sizeof(float));
  */

  MatMul(A, B, C);

  for (int i = 0; i < MAX * MAX; i++) {
    if ((i + 1) % MAX == 0) {
      printf("%.1lf\n", C[i]);
    }
    else {
      printf("%.1lf ", C[i]);
    }
  }

}



//int main() {
//  double A[MAX * MAX], B[MAX * MAX], C[MAX * MAX];
//  float time, time_avg = 0;
//  hipEvent_t start, stop;
//
//  hipEventCreate(&start); // create start event
//  hipEventCreate(&stop); // create stop event
//  hipEventRecord(start, 0); // record start event
//
//  for (int i = 0; i < MAX * MAX; i++) { // create A, B data
//    A[i] = 1.0;
//    B[i] = 1.0;
//    C[i] = 0.0;
//  }
//
//  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
//    double* cuda_A = 0;
//    double* cuda_B = 0;
//    double* cuda_C = 0;
//    hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(double));
//    hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(double));
//    hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(double));
//
//    hipMemcpy(cuda_A, A, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
//    hipMemcpy(cuda_B, B, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
//    //hipMemcpy(cuda_C, C, MAX * MAX * sizeof(double), hipMemcpyHostToDevice);
//
//    //dim3 dimGrid(MAX / BLOCKS, MAX / BLOCKS);
//    //dim3 dimBlock(BLOCKS, BLOCKS);
//
//    hipEventCreate(&start); // �ð� ���
//    hipEventCreate(&stop);
//    hipEventRecord(start, 0);
//
//    dim3 dimGrid(ceil((double)MAX / BLOCK_SIZE), ceil((double)MAX / BLOCK_SIZE));
//    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//
//    //matrix_multiplication_with_shared_memory << < dimGrid, dimBlock >> > (cuda_C, cuda_A, cuda_B); // matrix multiplication
//    matrix_multiplication_with_shared_memory << < dimGrid, dimBlock >> > (cuda_C, cuda_A, cuda_B, MAX, MAX, MAX); // matrix multiplication
//
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&time, start, stop);
//    hipEventDestroy(start);
//    hipEventDestroy(stop);
//
//    hipMemcpy(C, cuda_C, MAX * MAX * sizeof(double), hipMemcpyDeviceToHost);
//
//    if (t == 0) {
//      for (int i = 0; i < MAX * MAX; i++) {
//        if ((i + 1) % MAX == 0) {
//          printf("%.1lf\n", C[i]);
//        }
//        else {
//          printf("%.1lf ", C[i]);
//        }
//      }
//    }
//
//    time_avg += time;
//
//    // �޸� ����
//    hipFree(cuda_A);
//    hipFree(cuda_B);
//    hipFree(cuda_C);
//  }
//
//  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
//  return 0;
//}
