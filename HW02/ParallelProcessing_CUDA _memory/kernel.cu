#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <hipblas.h>
#include <hipblas.h>
// 32 64 128
#define MAX 8 // matrix length
#define BLOCKS 8
#define EPOCH 100
#pragma warning(disable:4996)

__global__ void matrix_multiplication_with_cublas(float* C, float* A, float* B) {
  int lda = MAX, ldb = MAX, ldc = MAX;
  const float alf = 1;
  const float bet = 0;
  const float* alpha = &alf;
  const float* beta = &bet;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, MAX, MAX, MAX, alpha, A, lda, B, ldb, beta, C, ldc);

  hipblasDestroy(handle);
}

int main() {
  float A[MAX * MAX], B[MAX * MAX], C[MAX * MAX];
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start); // create start event
  hipEventCreate(&stop); // create stop event
  hipEventRecord(start, 0); // record start event

  for (int i = 0; i < MAX * MAX; i++) { // create A, B data
    A[i] = 1;
    B[i] = 1;
    C[i] = 0;
  }

  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
    float* cuda_A = 0;
    float* cuda_B = 0;
    float* cuda_C = 0;
    hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(float));
    hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(float));
    hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(float));

    hipMemcpy(cuda_A, A, MAX * MAX * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, MAX * MAX * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(MAX / BLOCKS, MAX / BLOCKS);
    dim3 dimBlock(BLOCKS, BLOCKS);

    hipEventCreate(&start); // �ð� ���
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //matrix_multiplication_with_cublas(cuda_C, cuda_A, cuda_B); // matrix multiplication
    //matrix_multiplication_with_cublas << < dimGrid, dimBlock >> > (cuda_C, cuda_A, cuda_B); // matrix multiplication
    //hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, cuda_C, MAX * MAX * sizeof(float), hipMemcpyDeviceToHost);

    if (t == 0) {
      for (int i = 0; i < MAX * MAX; i++) {
        if ((i + 1) % MAX == 0) {
          printf("%.1lf\n", C[i]);
        }
        else {
          printf("%.1lf ", C[i]);
        }
      }
    }

    time_avg += time;

    // �޸� ����
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
  }

  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
  return 0;
}
