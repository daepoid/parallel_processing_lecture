#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
// 32 64 128
#define MAX 16
#define BLOCKS 32
#define EPOCH 100
#pragma warning(disable:4996)

__global__ void matrix_multiplication(int* C, int* A, int* B) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = 0; i < MAX; i++) {
    C[MAX * row + col] += A[MAX * row + i] * B[MAX * i + col];
  }
}

int main() {
  int A[MAX * MAX], B[MAX * MAX], C[MAX * MAX] = { 0, };
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start); // create start event
  hipEventCreate(&stop); // create stop event
  hipEventRecord(start, 0); // record start event

  for (int i = 0; i < MAX * MAX; i++) { // create A, B data
    A[i] = 1;
    B[i] = 1;
    C[i] = 0;
  }

  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
    int* cuda_A = 0;
    int* cuda_B = 0;
    int* cuda_C = 0;
    hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(int));
    hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(int));
    hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(int));

    hipMemcpy(cuda_A, A, MAX * MAX * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, MAX * MAX * sizeof(int), hipMemcpyHostToDevice);

    //dim3 dimGrid(MAX * MAX / BLOCKS);
    dim3 dimGrid(MAX * MAX / BLOCKS);
    dim3 dimBlock(BLOCKS, BLOCKS);

    hipEventCreate(&start); // �ð� ���
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //matrix_multiplication << < numBlocks, threadsPerBlock >> > (cuda_C, cuda_A, cuda_B); // Vector Addition ���
    matrix_multiplication << < dimGrid, dimBlock >> > (cuda_C, cuda_A, cuda_B); // Vector Addition ���
    //hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, cuda_C, MAX * MAX * sizeof(int), hipMemcpyDeviceToHost);

    if (t == 0) {
      for (int i = 0; i < MAX * MAX; i++) {
        if ((i + 1) % MAX == 0) {
          printf("%d\n", C[i]);
        }
        else {
          printf("%d ", C[i]);
        }
      }
    }

    time_avg += time;
    //printf("\n\n");

    // �޸� ����
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
  }

  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
  return 0;
}
