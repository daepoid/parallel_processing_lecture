#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
// 32 64 128
#define MAX 4
#define EPOCH 100
#define BLOCKS 8
#pragma warning(disable:4996)

__global__ void matrix_multiplication(int* C, int* A, int* B) {
  /*int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;*/

  int i = threadIdx.x;
  int j = blockIdx.x;

  for (int k = 0; k < MAX; k++) {
    C[MAX * j + i] += A[MAX * j + k] * B[MAX * k + i];
  }
}

int main() {
  int A[MAX * MAX], B[MAX * MAX], C[MAX * MAX] = { 0, };
  int cycle = MAX / 10;
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start); // create start event
  hipEventCreate(&stop); // create stop event
  hipEventRecord(start, 0); // record start event

  for (int i = 0; i < MAX * MAX; i++) { // create A, B data
    A[i] = 2;
    B[i] = 2;
  }

  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
    int* cuda_A = 0;
    int* cuda_B = 0;
    int* cuda_C = 0;
    hipMalloc((void**)&cuda_A, MAX * MAX * sizeof(int));
    hipMalloc((void**)&cuda_B, MAX * MAX * sizeof(int));
    hipMalloc((void**)&cuda_C, MAX * MAX * sizeof(int));

    hipMemcpy(cuda_A, A, MAX * MAX * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, MAX * MAX * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCKS, BLOCKS);
    dim3 numBlocks(MAX * MAX / threadsPerBlock.x);

    hipEventCreate(&start); // �ð� ���
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrix_multiplication << < numBlocks, threadsPerBlock >> > (cuda_C, cuda_A, cuda_B); // Vector Addition ���

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, cuda_C, MAX * MAX * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < MAX * MAX; i++) {
      if ((i + 1) % MAX == 0) {
        printf("%d\n", C[i]);
      }
      else {
        printf("%d ", C[i]);
      }
    }

    /*if (t == 0) {
      for (int i = 0; i < MAX; i++) {
        for (int j = 0; j < MAX; j++) {
          if (j == MAX - 1) {
            printf("%d\n", C[i * MAX + j]);
          }
          else {
            printf("%d ", C[i * MAX + j]);
          }
        }
      }
    }*/
    time_avg += time;
    //printf("\n\n");

    // �޸� ����
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);
  }

  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
  return 0;
}
