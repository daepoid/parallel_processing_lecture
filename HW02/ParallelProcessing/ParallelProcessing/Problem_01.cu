#include "hip/hip_runtime.h"
#pragma comment(lib,"cuda.lib")
#pragma comment(lib,"cudart.lib")

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
// 1024 4096 16384
#define MAX 16384
#define EPOCH 100
#define BLOCKS 8
#pragma warning(disable:4996)

__global__ void addVector(int* C, int* A, int* B) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
}

int main() {
  int A[MAX], B[MAX], C[MAX];
  int cycle = MAX / 10;
  float time, time_avg = 0;
  hipEvent_t start, stop;

  hipEventCreate(&start); // create start event
  hipEventCreate(&stop); // create stop event
  hipEventRecord(start, 0); // record start event

  for (int i = 0; i < MAX; i++) { // create A, B data
    A[i] = i;
    B[i] = i * i;
  }

  for (int t = 0; t < EPOCH; t++) { // EPOCH��ŭ �ݺ��Ͽ� ��սð��� ���
    int* cuda_A = 0;
    int* cuda_B = 0;
    int* cuda_C = 0;
    hipMalloc((void**)&cuda_A, MAX * sizeof(int));
    hipMalloc((void**)&cuda_B, MAX * sizeof(int));
    hipMalloc((void**)&cuda_C, MAX * sizeof(int));

    hipMemcpy(cuda_A, A, MAX * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, MAX * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCKS, BLOCKS);
    dim3 numBlocks(MAX / threadsPerBlock.x);

    hipEventCreate(&start); // �ð� ���
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    addVector << < numBlocks, threadsPerBlock >> > (cuda_C, cuda_A, cuda_B); // Vector Addition ���

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, cuda_C, MAX * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < MAX; i++) {
      if (i % cycle == 0) {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
      }
    }

    time_avg += time;
    printf("\n\n");
  }

  printf("time : %f\n", time_avg / (float)EPOCH); // ��� �ҿ� �ð� ���
  return 0;
}
